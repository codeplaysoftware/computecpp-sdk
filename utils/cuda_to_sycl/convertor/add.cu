
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
// specialize it based on type (specifically double) to avoid unaligned memory
// access compile errors
template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

// specialize for double to avoid unaligned memory
// access compile errors
template<>
struct SharedMemory<double>
{
    __device__ inline operator       double *()
    {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }

    __device__ inline operator const double *() const
    {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }
};

 
  __global__ void vecAdd(double *a, double *b, double *c, int n) {
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure we do not go out of bounds
    double *smem = SharedMemory<double>();
    if (id < n) {
      smem[threadIdx.x] = a[id] + b[id];
      c[id] = smem[threadIdx.x];
    }
  }
 
int main( int argc, char* argv[] )
{
    // Size of vectors
    int n = 1024;
 
    // Host input vectors
    double *h_a;
    double *h_b;
    //Host output vector
    double *h_c;
 
    // Device input vectors
    double *d_a;
    double *d_b;
    //Device output vector
    double *d_c;
 
    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(double);
 
    // Allocate memory for each vector on host
    h_a = (double*)malloc(bytes);
    h_b = (double*)malloc(bytes);
    h_c = (double*)malloc(bytes);
 
    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
 
    int i;
    // Initialize vectors on host
    for( i = 0; i < n; i++ ) {
        h_a[i] = sin(i)*sin(i);
        h_b[i] = cos(i)*cos(i);
    }
 
    // Copy host vectors to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
 
    int blockSize, gridSize;
 
    // Number of threads in each thread block
    blockSize = 256;
 
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);
    // shared memory size in byte
    int sharedmem = blockSize * sizeof(int);
 
    // Execute the kernel
    vecAdd<<<gridSize, blockSize, sharedmem>>>(d_a, d_b, d_c, n);
 
    // Copy array back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
 
    // Sum up vector c and print result divided by n, this should equal 1 within error
    double sum = 0;
    for(i=0; i<n; i++)
        sum += h_c[i];
    printf("final result: %f\n", sum/n);
 
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
 
    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);
 
    return 0;
}